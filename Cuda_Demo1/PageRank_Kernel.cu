#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "GPUTimer.h"
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <>  
#include "hip/device_functions.h"

namespace GraphRead_PageRank {

	//��ʼ��d_dangling_probs_sum
	__global__ void initialize_dangling_sum(double *d_dangling_probs_sum) {
		d_dangling_probs_sum[0] = 0.0;
	}

	//��ʼ��d_page_ranks_sum
	__global__ void initialize_pages_ranks_sum(double *d_page_ranks_sum) {
		d_page_ranks_sum[0] = 0.0;
	}

	__global__ void calculate_dangling_sum_and_normalize(Node *d_pages, double *d_page_probs, int pages_count, double *d_page_ranks_sum, double *d_dangling_probs_sum) {
		// Calculate page index from thread address
		// ���μ��ÿһ��thread
		int idx = blockIdx.y * blockDim.y + threadIdx.y;

		if (idx < pages_count) {
			d_page_probs[idx] /= d_page_ranks_sum[0];

			//������ǳ���Ϊ0�ĵ�
			if (d_pages[idx].dangling_node) {
				atomicAdd(d_dangling_probs_sum, double(d_page_probs[idx]));
			}
		}
	}

	__global__ void run_page_rank_iteration(Node *d_pages,
		double *d_page_probs,
		int *d_edges_list,  //����in_node
		int pages_count,
		double *d_pages_ranks_sum,
		double *d_dangling_probs_sum,
		double alpha) {

		//ȡ�߳��±�
		int idx = blockIdx.y * blockDim.y + threadIdx.y;

		double new_rank = 0.0;

		if (idx < pages_count) {
			double c_element = 0.0;


			//˭����idx��������d_edges_list�н��в���
			// d_page����ṹ��Ĵ�ŵ�start_idx��end_idx �൱���㷨��һ����cow_ptr
			int i_start = d_pages[idx].Adjstart_index;
			int i_end = d_pages[idx].Adjend_index;

			//���ھӽڵ㴫������ֵ���  
			// �����d_edges_list,���൱���㷨��һ���� col_idx
			for (int i = i_start; i < i_end; i++) {
				int from = d_edges_list[i];
				c_element += d_page_probs[from] / (d_pages[from].links_count);
			}

			//��alhpa���ӽ�������
			new_rank = ((1 - alpha) * 1.0 / pages_count) + (alpha * c_element);// +(alpha * 1.0 / pages_count * d_dangling_probs_sum[0]);  //����汾���������ҽڵ��Ӱ�죬Ҳ���԰��������һ��ɾ��
		}

		//__threadfence();
		__syncthreads();
		if (idx < pages_count) {
		d_page_probs[idx] = new_rank;  //���µ�pagerankֵ����probs���´ε�����
		//atomicExch(&d_page_probs[idx],new_rank);
		atomicAdd(d_pages_ranks_sum, new_rank);
		}
	}

	void Kernel::run_kernel() {
		// Calculate the grid and block sizes.
		// ceil ����ȥ��
		int grid_size = int(ceil(1.0 * nodes_count / MAX_BLOCK_SIZE));
		int block_size = MAX_BLOCK_SIZE;
		GPUTimer timer;
	
		// ��������þ���һ�����У�grid_size��block��ÿ��block����block_size��thread
		if (block_size < 1024) {
			dim3 dimGrid(1, grid_size);//ȷ��grid����block��
			dim3 dimBlock(1, block_size);//ȷ��block����thread��
			GPUTimer gpu_timer1;
			//gpu_timer1.e_start();
			timer.start();
			for (int i = 0; i < 100; ++i) {
				if (i > 0) {
					initialize_dangling_sum << <1, 1 >> > (d_dangling_probs_sum);
					calculate_dangling_sum_and_normalize << <dimGrid, dimBlock >> > (d_pages, d_pages_probs, nodes_count, d_pages_ranks_sum, d_dangling_probs_sum);
				}

				initialize_pages_ranks_sum << <1, 1 >> > (d_pages_ranks_sum);
				run_page_rank_iteration << < dimGrid, dimBlock >> > (d_pages, d_pages_probs, d_edges_list, nodes_count, d_pages_ranks_sum, d_dangling_probs_sum, ALPHA);
			}
			timer.stop();
			//	gpu_timer1.e_stop();
				//cout << "end PageRank time in gpu " << gpu_timer1.elapsed() << " ms." << endl;
		}

		printf("time :%lf\n", timer.elapsed());
		/*else {
			cout << "������һ��block�����ֵ1024" << endl;
		}*/
	}

	void Kernel::allocate_data(Node *pages, double *pages_probs, int *edges_list) {
		double one = 1.0, zero = 0.0;

		// Allocate memory at the gpu device
		hipMalloc((void **)&d_pages, sizeof(Node) * nodes_count);
		hipMalloc((void **)&d_pages_probs, sizeof(double) * nodes_count);
		hipMalloc((void **)&d_edges_list, sizeof(int) * edges_count);
		hipMalloc((void **)&d_pages_ranks_sum, sizeof(double));  //
		hipMalloc((void **)&d_dangling_probs_sum, sizeof(double));

		// Copy data from host (cpu) to the gpu
		hipMemcpy(d_pages, pages, sizeof(Node) * nodes_count, hipMemcpyHostToDevice);
		hipMemcpy(d_pages_probs, pages_probs, sizeof(double) * nodes_count, hipMemcpyHostToDevice);
		hipMemcpy(d_edges_list, edges_list, sizeof(int) * edges_count, hipMemcpyHostToDevice);
		hipMemcpy(d_pages_ranks_sum, &one, sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_dangling_probs_sum, &zero, sizeof(double), hipMemcpyHostToDevice);
	}

	double *Kernel::get_result() {
		double *pages_probs = new double[nodes_count];

		hipMemcpy(pages_probs, d_pages_probs, sizeof(double) * nodes_count, hipMemcpyDeviceToHost);
		hipError_t e = hipGetLastError();
		if (e != hipSuccess) {
			printf("CUDA����ʧ��  %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
			exit(0);
		}

		return pages_probs;
	}

} /* namespace PageRank */
