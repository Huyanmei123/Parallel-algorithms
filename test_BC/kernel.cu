#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include<math.h>
#include <string>
#include <vector>
#include <fstream>
#include <windows.h>
#include <time.h>
#include <iomanip>
#include <sstream>
#include <>  //����blockIdx,threadIdx,gridDim
#include "hip/device_functions.h"
using namespace std;

struct check {
	int edge1;
	int edge2;
	int node;
};
#define THREAD_COUNT 1024 

//���Լ����Կ����Դ�2G,��������ʱ����2G
#define MAX_MEMORY ((long long)12e9)

//��ӡ��ȡGPU�豸������Ϣ
void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//��ʼ��CUDA
bool InitCUDA()
{
	int count;

	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;

	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//��ӡ�豸��Ϣ
		printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

/*
�ȶ�ȡ���ڽӱ�
��ת����SCR�洢�ṹ�洢����
row_ptr:�ڵ��
col_ind:�ڵ��ڽӱ�
ֱ�Ӷ�ȡ��GPU��host�Դ���
*/
void read_graph(std::string fname, int *&row_ptr, int *&col_ind, int &num_nodes, int &num_edges, bool zero_based = false)
{
	std::ifstream input(fname.c_str());
	if (input.fail())
		throw "No file is found in the current path!";

	// �����磬����scrģʽ���������ַ����ȴ����
	std::string line = "%";
	while (line.find("%") != std::string::npos)
	{
		getline(input, line);
	}
	
	std::istringstream ss(line.c_str());
	ss >> num_nodes >> num_nodes >> num_edges;
	int edge_cnt = 0;
	int v1, v2;
	//struct check *c=new check[num_edges];
	//�ڽӱ�
	std::vector< std::vector<int> > adj_list(num_nodes);
	for (int i = 0; i < num_edges; i++)
	{
		getline(input, line);
		std::istringstream inp(line.c_str());
		inp >> v1 >> v2;
	/*	c[i].edge1 = v1;
		c[i].edge2 = v2;*/
		if (v1 != v2)
		{
			adj_list[v1].push_back(v2); // add the edge v1->v2
			adj_list[v2].push_back(v1); // add the edge v2->v1
			edge_cnt++;
		}
	}
	input.close();
	num_edges = edge_cnt;


	//gpu host�������ڴ�
	//+1����Ϊһ��ʼ����Ҫ���������degree
	hipHostMalloc((void **)&row_ptr, sizeof(int) * (num_nodes + 1));
	hipHostMalloc((void **)&col_ind, sizeof(int) * (2 * num_edges));

	//�ڽӱ�תSCR�ṹ
	row_ptr[0] = 0;  //��һ��Ϊ0
	int index = 0;
	for (int v = 0; v < num_nodes; v++)
	{
		row_ptr[v + 1] = adj_list[v].size();
		for (int i = 0; i < (int)adj_list[v].size(); i++)
		{
			col_ind[index] = adj_list[v][i];
			index++;
		}
	}

	//����Ҫ�ۼ�һ�²ŵõ����յ�row_ptr
	for (int v = 1; v < num_nodes + 1; v++)
	{ // cumulative sum
		row_ptr[v] += row_ptr[v - 1];
	}

}

__global__
void cent_kernel(double *results,  int *dist,  double *sigma,  double *delta, int *rp, int *ci, int n) {//rp:row_ptr ci:cow_idx
	//�����ڴ����
	__shared__ int level;
	__shared__ int visited;
	__shared__ double dist_sum;
	//__shared__ double dist_sum_2;
	for (int s = blockIdx.x; s < n; s += gridDim.x) {

		if (threadIdx.x == 0) {
			//results[s] = rp[s + 1] - rp[s]; // ����������� degree 1 
			level = 0;
			dist_sum = 0.000;
			visited = 1;
		//	dist_sum_2 = 0.0;
			dist[blockIdx.x * n + s] = 0;
			sigma[blockIdx.x * n + s] = 1; //sigma��Ҫ��ǰ��ֵΪ1
		}

		__syncthreads();

		// BFS���ȿ���Ѱ��dist
		while (visited == 1) {
			if (threadIdx.x == 0) visited = 0;
			for (int node = threadIdx.x; node < n; node += blockDim.x) {
				for (int edge = rp[node]; edge < rp[node + 1]; edge++) { //�൱�ڱ���cd_idx
					int &adj = ci[edge]; //ȡ����

					if (dist[(blockIdx.x * n) + adj] == level && dist[(blockIdx.x * n) + node] == -1){
						dist[(blockIdx.x * n) + node] = level + 1;
						visited = 1;
					}

				    if (dist[(blockIdx.x * n) + adj] == level && dist[(blockIdx.x * n) + node] == level + 1) {				
						
						atomicAdd(&sigma[(blockIdx.x * n) + node],sigma[(blockIdx.x * n) + adj]);

					
					}

				}
			
			}
			__syncthreads();
			if (threadIdx.x == 0) level++;   //level�൱��brandes���d,��������ÿ����1
			__syncthreads();
		}
		
	
	
		int dist2_cnt = 0;
	
		// �����ۼ�
		if (threadIdx.x == 0) {
			for (int i = 0; i < n; i++) {
				if (dist[(blockIdx.x * n) + i] > 0) {					

					dist_sum += dist[(blockIdx.x * n) + i];
					
				}
		
			}
				results[2 * n + s] = (n-1)/dist_sum; //   ����closenes
		}
		
		
		//�������betweeness
		while (level >0) {
			for (int node = threadIdx.x; node < n; node += blockDim.x) {
				if (dist[blockIdx.x * n + node] == level) {   //�Ӿ�����Զ�ĵ㿪ʼ�Ӻ���ǰ
					for (int edge = rp[node]; edge < rp[node + 1]; edge++) {
						int adj = ci[edge];
						if (dist[(blockIdx.x * n) + adj] + 1 == dist[(blockIdx.x * n) + node]) {
							//betweeness��ԭ�Ӽӣ�����ͬ������
							atomicAdd(&delta[(blockIdx.x * n) + adj], (sigma[(blockIdx.x * n) + adj] * 1.0) / sigma[(blockIdx.x * n) + node] * (1 + delta[(blockIdx.x * n) + node]));
						}
					}
				 	atomicAdd(&results[3 * n + node], delta[(blockIdx.x * n) + node]);
				}
	
			}
			__syncthreads();
			if (threadIdx.x == 0) level--;
			__syncthreads();
		}

		//ÿ���������������ֵ���Ա���һ���ڵ���
		for (int i = 0; i < n; i++) {
			dist[(blockIdx.x * n) + i] = -1;
			sigma[(blockIdx.x * n) + i] = 0;
			delta[(blockIdx.x * n) + i] = 0;
		}
		__syncthreads();
	}

}





double* compute_centralities(int *rp, int *ci, int n, float &time_taken) {
	printf("node_number: %d \n", n);
	/*
		block_count ��������ʵû���ر�׼ȷ�Ĺ�ʽ��һ����˵������������Ƕ��٣�����100000���㣬�����ÿ��block������THREAD_COUNT����Ϊ1024,100000/1024�Ϳ���
	*/

	const int BLOCK_COUNT =512; //����ֵ��һ��Ϊ128 �� 256

	printf("block_count �п���Ϊ:%d \n", BLOCK_COUNT);
	double *sigma;
	int  *dist;
	double *delta;
	double *d_results;

	hipMalloc((void **)&d_results, sizeof(double) * n * 4); //4�������ԣ�n���ڵ�
	hipMalloc((void **)&sigma, sizeof(double  ) * n * BLOCK_COUNT);  //*BLOCK_COUNT,ÿ��block�е�������
	hipMalloc((void **)&dist, sizeof( int) * n * BLOCK_COUNT); //
	hipMalloc((void **)&delta, sizeof(double) * n * BLOCK_COUNT);

	hipMemset(dist, -1, sizeof( int) * n * BLOCK_COUNT);//��ʼ��
	hipMemset(sigma, 0, sizeof(double ) * n * BLOCK_COUNT);
	hipMemset(delta, 0, sizeof(double) * n * BLOCK_COUNT);
	hipMemset(d_results, 0, sizeof(double) * 4 * n);

	hipEvent_t start, end; //��¼�¼�
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

   cent_kernel << <BLOCK_COUNT, THREAD_COUNT >> > (d_results, dist, sigma, delta, rp, ci, n);
	//cent_kernel_seq << <BLOCK_COUNT, THREAD_COUNT >> > (d_results, dist, sigma, delta, rp, ci, n);
	hipDeviceSynchronize(); //��ͬ������ʱ�����

	hipEventRecord(end);
	hipEventSynchronize(end);//��ͬ������ʱ�����

	hipEventElapsedTime(&time_taken, start, end);//��¼gpuʱ��


	double *results;

	hipHostMalloc((void **)&results, sizeof(double) * n * 4);
	hipMemcpy(results, d_results, sizeof(double) * n * 4, hipMemcpyDeviceToHost);

	hipFree(sigma);
	hipFree(dist);
	hipFree(delta);
	hipFree(d_results);


	hipDeviceSynchronize();//���ж�����ͬ�����ڷ���ֵ
	return results;
}


int main()
{
	//CUDA ��ʼ�����鿴GPU��������
	if (!InitCUDA()) {
		return 0;
	}

	/*
	���ݸ�ʽ��飺1.�����Ƿ���ڹ����㣬�����Ƚ��д��� 2.���ݵĵ�һ���Ƿ�������� node_num>>node_num>>edge_num
	����ǰ��飺��1�������ļ����滻  ��2�� д���ļ����滻
	*/
	
	std::string filename = "D:/��˼Դ/data/musae_facebook_edges-22470-171002.txt";
	int *row_ptr, *col_ind;
	int num_nodes, num_edges;
	read_graph(filename, row_ptr, col_ind, num_nodes, num_edges);
	printf("Graph is read: %s.\n", filename.c_str());

	printf("done scr storage!!!!!\n");
	for (int j = 0; j < num_nodes + 1; j++)
	{
		//printf("row_ptr: %d \n", row_ptr[j]);
	}

	float time_taken;
	int *rp;
	int *ci;
	
	hipMalloc((void **)&rp, sizeof(int) * (num_nodes + 1));
	hipMalloc((void **)&ci, sizeof(int) * (num_edges * 2));

	printf("CUDA memory parameters are allocated for kernel function.\n");

	//SCR�洢�ṹ-->��host�豸���Ƶ�device�豸��
	hipMemcpy(rp, row_ptr, sizeof(int) * (num_nodes + 1), hipMemcpyHostToDevice);
	hipMemcpy(ci, col_ind, sizeof(int) * (num_edges * 2), hipMemcpyHostToDevice);
	printf("CUDA memory parameters are set for kernel function.\n");

	double *results = compute_centralities(rp, ci, num_nodes, time_taken);

	printf("Kernel function is finished.�˺���ִ����ɡ�\n");

	printf("Centrality Results:\n");
	printf("------------------------------------------------------------------------\n");
	ofstream write;
	write.open("D:\\��˼Դ\\result\\BC\\com-amazon\\com-amazon.ungraph-334863-bc-v2.txt");
	ofstream write1;
	write1.open("D:\\��˼Դ\\result\\CC\\com-amazon\\com-amazon.ungraph-334863-cc-v2.txt");
	//double nom = (num_nodes - 1)*(num_nodes - 2);
	for (int i = 0; i < num_nodes; i++)
	{
		// printf("%.5f; %.5f; %.5f; %.5f\n ", results[i], results[num_nodes + i], results[2 * num_nodes + i], results[3 * num_nodes + i]);
	
	  //  write << fixed << setprecision(2) << results[3 * num_nodes + i]/2<< endl;
		//write1 << fixed << setprecision(6) << results[2 * num_nodes + i] << endl;
		
	 cout << i<<": "<<" cc:" << results[2 * num_nodes + i]<<"    bc: "<<fixed << setprecision(5)<< results[3 * num_nodes + i]/2 << endl;
	// cout << fixed << setprecision(3) << results[2 * num_nodes + i] << endl;
			//cout << i << ": " << " cc:" << results[2 * num_nodes + i] << endl;
		
	

	}
	write.close();

	printf("the time use of Kernel function: %.5f ms\n", time_taken);

	hipHostFree(results);
	hipHostFree(row_ptr);
	hipHostFree(col_ind);
	hipFree(rp);
	hipFree(ci);

	system("pause");
	return 0;
}